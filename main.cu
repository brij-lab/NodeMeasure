/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

/**
 * @author Brij Mohan Lal Srivastava
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>

static const char DELIMITER = ' ';

typedef struct node node;
struct node {
	int nodenum;
	int fidparts[68][2];
	int pose;
	int nfid;
};

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

/**
 * CUDA kernel function that calculates minsum of nodes
 */

__global__ void sum(int * nodeData, float * sum, int * combs, int * pose, int * filters, int * cparts, int nodeCount, int combsCount, int dataCount) {
	int combIdx = blockIdx.x *blockDim.x + threadIdx.x;

	//printf("Thread no. : %d\n", combIdx);
	if (combIdx < combsCount * 2 - 2) {
		//printf("pass 1\n");
		int node1Idx = combs[combIdx * 2];
		int node2Idx = combs[combIdx * 2 + 1];

		printf("Node indexes %d, %d ... \n", node1Idx, node2Idx);

		int node1startIdx = node1Idx * dataCount;
		int node2startIdx = node2Idx * dataCount;

		int node1pose = pose[node1Idx];
		int node2pose = pose[node2Idx];

		//printf("pass2\n");
		if(abs(node1pose - node2pose) > 3) {
			//printf("pass3\n");
			sum[combIdx] = -1;
		}
		else
		{
			//printf("pass4\n");
			int i, j, k;
			int node1data[68][2], node2data[68][2], node1fdata[99][2], node2fdata[99][2];

			int cnt = 0, start = node1startIdx, end = node1startIdx + dataCount;
			for (i = start; i < end; i+=2) {
				node1data[cnt][0] = nodeData[i];
				node1data[cnt][1] = nodeData[i + 1];
				cnt++;
			}

			cnt = 0; start = node2startIdx; end = node2startIdx + dataCount;
			for (i = start; i < end; i+=2) {
				node2data[cnt][0] = nodeData[i];
				node2data[cnt][1] = nodeData[i + 1];
				cnt++;
			}

			int node1posedata[68], node2posedata[68];

			cnt = 0; start = node1pose * 68; end = node1pose * 68 + 68;
			for (i = start; i < end; i++) {
				node1posedata[cnt] = filters[i];
				cnt++;
			}
			cnt = 0; start = node2pose * 68; end = node2pose * 68 + 68;
			for (i = start; i < end; i++) {
				node2posedata[cnt] = filters[i];
				cnt++;
			}

			// Re-organise node data
			for (i = 0; i < 68; i++) {
				if (node1posedata[i] != -1) {
					node1fdata[node1posedata[i]][0] = node1data[i][0];
					node1fdata[node1posedata[i]][1] = node1data[i][1];
				}
			}
			for (i = 0; i < 68; i++) {
				if (node2posedata[i] != -1) {
					node2fdata[node2posedata[i]][0] = node2data[i][0];
					node2fdata[node2posedata[i]][1] = node2data[i][1];
				}
			}

			// Match and calculate sum
			int pose1, pose2;
			if(node1pose < node2pose) {
				pose1 = node1pose;
				pose2 = node2pose;
			}
			else
			{
				pose1 = node2pose;
				pose2 = node1pose;
			}

			int cpIdx;
			if (pose1 < 11) {
				cpIdx = ((4 * (pose1 - 1))  + (pose2 - pose1)) * 68;

			}
			else
			{
				if (pose1 == 11) {
					cpIdx = 68 * (40 + pose2 - pose1);
				}
				else if (pose1 == 12) {
					cpIdx = 68 * (43 + pose2 - pose1);
				}
				else
				{
					cpIdx = 68 * 45;
				}
			}

			int ncparts = 0;
			while(cparts[cpIdx] != -1 && ncparts < 68) {
				ncparts++;
			}

			int commonp[68];
			int ncpIdx = 0;
			for (i = cpIdx; i < cpIdx + 68; i++) {
				commonp[ncpIdx] = cparts[i];
				ncpIdx++;
			}

			float min = FLT_MAX;
			float csum;
			// i, j for local area survey
			for (i = -4; i < 5; i++) {
				for (j = -4; j < 5; j++) {

					csum = 0.0;
					// k for matching only common parts
					for (k = 0; k < ncparts; k++) {
						int x1 = node1fdata[commonp[k]][0] + i;
						int x2 = node2fdata[commonp[k]][0];

						int y1 = node1fdata[k][1] + j;
						int y2 = node2fdata[k][1];

						csum += ((x1 - x2) * (x1 - x2)) + ((y1 - y2) * (y1 - y2));

					}

					csum = sqrtf(csum) / ncparts;
					min = (csum < min) ? csum : min;
				}
			}

			sum[combIdx] = min;
		}
	}
}

/**
 * Util function to split up the string into tokens
 */
char** str_split(char* a_str, const char a_delim) {
	char** result = 0;
	size_t count = 0;
	char* tmp = a_str;
	char* last_comma = 0;
	char delim[2];
	delim[0] = a_delim;
	delim[1] = 0;

	/* Count how many elements will be extracted. */
	while (*tmp) {
		if (a_delim == *tmp) {
			count++;
			last_comma = tmp;
		}
		tmp++;
	}

	/* Add space for trailing token. */
	count += last_comma < (a_str + strlen(a_str) - 1);

	/* Add space for terminating null string so caller
	 knows where the list of returned strings ends. */
	count++;

	if(result) {
		free(result);
	}
	result = (char **) malloc(sizeof(char *) * count);

	if (result) {
		size_t idx = 0;
		char* token = strtok(a_str, delim);

		while (token) {
			*(result + idx++) = strdup(token);
			token = strtok(0, delim);
		}
		*(result + idx) = 0;
	}

	return result;
}

/**
 * Util to calculate nCr combinations
 */
int nCr(int n, int r) {
	if(r > n / 2) r = n - r; // because C(n, r) == C(n, n - r)
	long long ans = 1;
	int i;

	for(i = 1; i <= r; i++) {
		ans *= n - r + i;
		ans /= i;
	}

	return ans;
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char **argv) {

	for (int i = 0; i < argc; ++i)
	{
		printf("argv[%d]: %s\n", i, argv[i]);
	}

	int i, j;
	char line[80];
	char ** tokens;
	FILE * fr;
	FILE * fFilters;
	FILE * fCommon;
	int NODE_COUNT;
	node ** nodes;
	int * pose;
	bool meta = true;
	bool first = true;
	int dataidx;
	int nodeidx = 0;
	int x, y, w, h;

	printf("Started ... \n");

	printf("Reading bounding boxes ... \n");
	// Read input
	fr = fopen("/home/brij/Downloads/bounding_boxes.txt", "rt");

	while (fgets(line, 80, fr) != NULL) {
		//printf("line = %s\n", line);
		if (first) {
			sscanf(line, "%d", &NODE_COUNT);
			//printf("1 : %d\n", NODE_COUNT);
			nodes = (node **) malloc(sizeof(node *) * NODE_COUNT);
			pose = (int *) malloc(sizeof(int) * NODE_COUNT);
			first = false;
		} else if (!first && meta) {
			//printf("2\n");
			nodes[nodeidx] = (node *) malloc(sizeof(node));

			strtok(line, "\n");
			tokens = str_split(line, DELIMITER);
			if (tokens) {

				sscanf(*(tokens + 0), "%d", &(nodes[nodeidx]->nodenum));
				sscanf(*(tokens + 1), "%d", &(nodes[nodeidx]->pose));
				sscanf(*(tokens + 2), "%d", &(nodes[nodeidx]->nfid));

				pose[nodeidx] = nodes[nodeidx]->pose;
				for (i = 0; *(tokens + i); i++)
				{
					//printf("month=[%s]\n", *(tokens + i));
					free(*(tokens + i));
				}
				free(tokens);

				//printf("%d, %d, %d\n", nodes[nodeidx]->nodenum, nodes[nodeidx]->pose, nodes[nodeidx]->nfid);
				dataidx = 0;
				//nodes[nodeidx]->fidparts = (int **)malloc(sizeof(int *) * 68 * 2);
				memset(nodes[nodeidx]->fidparts, 0, sizeof(nodes[nodeidx]->fidparts[0][0]) * 68 * 2);
			}
			meta = false;

		} else {
			//printf("3\n");
			strtok(line, "\n");
			tokens = str_split(line, DELIMITER);

			if (tokens) {
				//printf("Printing tokens...\n");
				sscanf(*(tokens + 0), "%d", &x);
				sscanf(*(tokens + 1), "%d", &y);
				sscanf(*(tokens + 2), "%d", &w);
				sscanf(*(tokens + 3), "%d", &h);

				for (i = 0; *(tokens + i); i++)
				{
					//printf("month=[%s]\n", *(tokens + i));
					free(*(tokens + i));
				}
				free(tokens);

				//printf("%d, %d, %d, %d\n", x, y, w, h);
			}
			//printf("4\n");

			//nodes[nodeidx]->fidparts[dataidx] = (int *) malloc(sizeof(int) * 2);
			nodes[nodeidx]->fidparts[dataidx][0] = x + w / 2;
			nodes[nodeidx]->fidparts[dataidx][1] = y + h / 2;

			dataidx++;


			//printf("data idx : %d\n", dataidx);
			if (dataidx == nodes[nodeidx]->nfid) {
				meta = true;
				nodeidx++;
			}

		}
	}

	printf("Reading filter ids ... \n");
	fFilters = fopen("/home/brij/Downloads/filter_ids.txt", "rt");

	int * filter = (int *) malloc(sizeof(int) * 68 * 13);
	int filIdx = 0;
	meta = true;
	int dataCnt = 0, filpoints = 0, tofill = 0, temp;
	while (fgets(line, 80, fFilters) != NULL) {
		if (meta) {
			strtok(line, "\n");
			tokens = str_split(line, DELIMITER);

			if (tokens) {
				sscanf(*(tokens + 1), "%d", &dataCnt);
				filpoints = dataCnt;

				for (i = 0; *(tokens + i); i++)
				{
					free(*(tokens + i));
				}
				free(tokens);
			}
			meta = false;
		}
		else
		{
			sscanf(line, "%d", &temp);
			filter[filIdx] = temp - 1; // To account for 1-indexing in matlab (Thanks to mallik)
			dataCnt--;
			filIdx++;

			if (dataCnt == 0) {
				meta = true;
				if (filpoints < 68) {
					tofill = 68 - filpoints;
					for (i = 0; i < tofill; i++) {
						filter[filIdx] = -1;
						filIdx++;
					}
				}
			}
		}
	}

	fclose(fFilters);

	printf("Reading common parts ... \n");
	fCommon = fopen("/home/brij/Downloads/common_parts.txt", "rt");

	int * cparts = (int *) malloc(sizeof(int) * 46 * 68);
	meta = true; filIdx = 0;
	while (fgets(line, 80, fCommon) != NULL) {
		if (meta) {
			strtok(line, "\n");
			tokens = str_split(line, DELIMITER);

			if (tokens) {
				sscanf(*(tokens + 2), "%d", &dataCnt);
				filpoints = dataCnt;

				for (i = 0; *(tokens + i); i++)
				{
					free(*(tokens + i));
				}
				free(tokens);
			}
			meta = false;
		}
		else
		{
			sscanf(line, "%d", &temp);
			cparts[filIdx] = temp - 1; // To account for 1-indexing in matlab (Thanks to mallik)
			dataCnt--;
			filIdx++;

			if (dataCnt == 0) {
				meta = true;
				if (filpoints < 68) {
					tofill = 68 - filpoints;
					for (i = 0; i < tofill; i++) {
						cparts[filIdx] = -1;
						filIdx++;
					}
				}
			}
		}
	}

	fclose(fCommon);

	//for (i = 0; i < 68*13; i++) {
		//printf("fil : %d\n", filter[i]);
	//}

	int combCount = nCr(NODE_COUNT, 2);
	int * combs = (int *) malloc(sizeof(int) * combCount * 2);
	int combIdx = 0;
	for (i = 0; i < NODE_COUNT - 1; i++) {
		for (j = i + 1; j < NODE_COUNT; j++) {
			combs[combIdx] = i;
			combs[combIdx + 1] = j;
			combIdx += 2;
		}
	}

	//printf("combs = %d, last comb index = %d\n", combCount, combIdx);
	/*
	for (i = 0; i < combCount * 2; i+=2) {
		printf("%d, %d\n", combs[i], combs[i + 1]);
	}
	*/

	printf("Nodes = %d\n", NODE_COUNT);
	// Flatten 3-d array
	int arrSize = sizeof(int) * NODE_COUNT * 68 * 2;
	int * nodeData = (int *) malloc(arrSize);

	for (i = 0; i < NODE_COUNT; i++) {
		for (j = 0; j < 68; j++) {
			nodeData[(i * 68 * 2) + (j * 2) + 0] = nodes[i]->fidparts[j][0];
			nodeData[(i * 68 * 2) + (j * 2) + 1] = nodes[i]->fidparts[j][1];
		}
	}

	printf("Loading data into GPU ... \n");

	// Nodes size
	int * d_nodeData;
	int * d_combs;
	float * h_sums;
	float * d_sums;
	int * d_pose;
	int * d_filters;
	int * d_cparts;

	h_sums = (float *) malloc(sizeof(float) * combCount);

	CUDA_CHECK_RETURN(hipMalloc(&d_nodeData, arrSize));
	CUDA_CHECK_RETURN(hipMalloc(&d_sums, sizeof(float) * combCount));
	CUDA_CHECK_RETURN(hipMalloc(&d_combs, sizeof(int) * combCount * 2));
	CUDA_CHECK_RETURN(hipMalloc(&d_pose, sizeof(int) * NODE_COUNT));
	CUDA_CHECK_RETURN(hipMalloc(&d_filters, sizeof(int) * 68 * 13));
	CUDA_CHECK_RETURN(hipMalloc(&d_cparts, sizeof(int) * 68 * 46));
	CUDA_CHECK_RETURN(hipMemcpy(d_nodeData, nodeData, arrSize, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_combs, combs, sizeof(int) * combCount * 2, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_pose, pose, sizeof(int) * NODE_COUNT, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_filters, filter, sizeof(int) * 68 * 13, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_cparts, cparts, sizeof(int) * 68 * 46, hipMemcpyHostToDevice));

	int gridSize, threads;

	printf("Combination count = %d \n", combCount);
	if (combCount < 1000) {
		gridSize = 1;
		threads = combCount;
	}
	else
	{
		gridSize = (combCount % 1000 == 0) ? combCount / 1000 : combCount / 1000 + 1;
		threads = 1000;
	}

	printf("Launching kernel gridsize = %d, threads = %d... \n", gridSize, threads);
	sum<<<gridSize, threads>>> (d_nodeData, d_sums, d_combs, d_pose, d_filters, d_cparts, NODE_COUNT, combCount, 68 * 2);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(h_sums, d_sums, sizeof(float) * combCount, hipMemcpyDeviceToHost));


	printf("Printing result ... \n");
	for (i = 0; i < combCount; i++) {
		printf("Sum %d = %f\n", i, h_sums[i]);
	}

	CUDA_CHECK_RETURN(hipFree((void* ) d_nodeData));
	CUDA_CHECK_RETURN(hipFree((void* ) d_combs));
	CUDA_CHECK_RETURN(hipFree((void* ) d_sums));
	CUDA_CHECK_RETURN(hipFree((void* ) d_pose));
	CUDA_CHECK_RETURN(hipFree((void* ) d_filters));
	CUDA_CHECK_RETURN(hipFree((void* ) d_cparts));
	CUDA_CHECK_RETURN(hipDeviceReset());

	fclose(fr);
	return 0;
}
